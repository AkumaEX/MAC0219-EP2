#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "structure.h"
#include "reduction.h"

inline
hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess)
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    return result;
}

int main(int argc, char *argv[]) {
	int i, j, k;

    if (argc != 2) {
        printf("Uso: %s <caminho_lista_matrizes>\n", argv[0]);
        return 3;
    }

    char *filepath = argv[1];

    int num_matrices;
    int **structure_h = load_structure(filepath, &num_matrices);
	int BlockSize = 64;
	int GridSize = (num_matrices+BlockSize-1)/BlockSize;

    int *structure_d[9];
	int *result_d[9];
	int *result_h[9];
	for(i = 0; i < 9; i++)
		result_h[i] = new int[GridSize];
	
	hipSetDevice(0);
	for(i = 0; i < 9; i ++) {
		checkCuda(hipMalloc((void**) &structure_d[i], num_matrices * sizeof(int)));
		checkCuda(hipMalloc((void**) &result_d[i], GridSize * sizeof(int)));

		checkCuda(hipMemcpy(structure_d[i], structure_h[i], num_matrices * sizeof(int), hipMemcpyHostToDevice));
		reduction<<<GridSize, BlockSize, BlockSize>>>(structure_d[i], result_d[i], num_matrices);
		checkCuda(hipMemcpy(result_h[i], result_d[i], GridSize  * sizeof(int), hipMemcpyDeviceToHost));
		
	}
	//hipDeviceSynchronize();

	//for(i = 0; i < 9; i++)

	int final[3][3];

	for(i = 0; i < 3; i++) {
		for(j = 0; j < 3; j++) {
			int min = result_h[i*3+j][0];
			for(k = 1; k < GridSize; k++) {
				if(min > result_h[i*3+j][k]) min = result_h[i*3+j][k];
			}
			final[i][j] = min;
		}
	}

    print_matrix(final);
	
	for(i = 0; i < 9; i++) {
		free(structure_h[i]);
		delete [] result_h[i];
		checkCuda(hipFree(structure_d[i]));
		checkCuda(hipFree(result_d[i]));
	}
    free(structure_h);

    return EXIT_SUCCESS;
}
