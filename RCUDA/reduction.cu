#include "hip/hip_runtime.h"
//Caio Henrique Silva Ramos - NUSP 9292991
//Julio Kenji Ueda - NUSP 9298281

#include <stdio.h>
#include <limits.h>
#include "reduction.h"

__global__ void reduction(int *structure, int *result, int N) {

	//Shared vector for min's
    extern __shared__ int sdata[];  
	int i = threadIdx.x;
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	
	int min = INT_MAX;
	while(tid < N) {
		min = imin(min, structure[tid]);
		tid += blockDim.x*gridDim.x;
	}

	sdata[i] = min;	
	__syncthreads();

	//Compute the min's
	int s = blockDim.x/2;
	while(s != 0) {
		if(i < s) {
			sdata[i] = imin(sdata[i], sdata[i+s]);
		}

		__syncthreads();
		s /= 2;
	}
	//Save the results
	if(i == 0) result[blockIdx.x] = sdata[0];
}
