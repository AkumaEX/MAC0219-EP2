#include "hip/hip_runtime.h"
#include <stdio.h>
#include "reduction.h"

__global__ void reduction(int *structure, int num_matrices, int task_size, int *result) {

    extern __shared__ int sdata[];  // guarda o minimo que uma thread consegue encontrar
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int start = (tid * task_size) + (num_matrices * bid);  // indice da estrutura que a thread comeca trabalhar
    int finish = (start + task_size < num_matrices * (bid + 1)) ? start + task_size : num_matrices * (bid + 1); // indice da estrutura onde a thread precisa parar

    sdata[tid] = structure[start]; // o menor elemento inicial e o primeiro elemento
    __syncthreads();

    int i; // cada thread percorre os elementos e vai guardando o minimo no indice da sua thread
    for (i = start + 1; i < finish; i++)
        sdata[tid] = sdata[tid] ^ ((structure[i] ^ sdata[tid]) & -(structure[i] < sdata[tid]));
    __syncthreads();

    if (tid == 0) { // a thread zero sozinha percorre todos os elementos minimos e devolve o menor deles
        for (i = 1; i < blockDim.x; i++)
            sdata[0] = sdata[0] ^ ((sdata[i] ^ sdata[0]) & -(sdata[i] < sdata[0]));
        result[bid] = sdata[0];
    }
}