#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include "reduction.h"

__global__ void reduction(int *structure, int *result, int N) {

    extern __shared__ int sdata[];  
	int i = threadIdx.x;
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	
	int min = INT_MAX;
	while(tid < N) {
		min = imin(min, structure[tid]);
		tid += blockDim.x*gridDim.x;
	}

	sdata[i] = min;	
	__syncthreads();

	int s = blockDim.x/2;
	while(s != 0) {
		if(i < s) {
			sdata[i] = imin(sdata[i], sdata[i+s]);
		}

		__syncthreads();
		s /= 2;
	}

	if(i == 0) result[blockIdx.x] = sdata[0];
}
