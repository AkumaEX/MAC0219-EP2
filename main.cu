#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "structure.h"
#include "reduction.h"

#define GRIDDIM 9
#define BLOCKDIM 64

int main(int argc, char *argv[]) {

    if (argc != 2) {
        printf("Uso: %s <caminho_lista_matrizes>\n", argv[0]);
        return 3;
    }

    char *filepath = argv[1];

    int num_matrices;
    int *structure_h = load_structure(filepath, &num_matrices);
    int *structure_d;
    int result_h[9];
    int *result_d;
    int num_threads = (int) ceil(num_matrices / 2);
    int block_dim = (num_threads > BLOCKDIM) ? BLOCKDIM : num_threads;
    int task_size = (int) ceil(num_matrices / block_dim); // numero de elementos que a thread precisa comparar
    printf("num_threads: %d, block_dim: %d, task_size: %d\n", num_threads, block_dim, task_size);
    hipMalloc((void**) &structure_d, 9 * num_matrices * sizeof(int));
    hipMalloc((void**) &result_d, 9 * sizeof(int));

    hipMemcpy(structure_d, structure_h, 9 * num_matrices * sizeof(int), hipMemcpyHostToDevice);
    reduction<<<GRIDDIM, block_dim, block_dim>>>(structure_d, num_matrices, task_size, result_d);
    hipMemcpy(result_h, result_d, 9 * sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(result_h);

    free(structure_h);
    hipFree(structure_d);
    hipFree(result_d);

    return EXIT_SUCCESS;
}